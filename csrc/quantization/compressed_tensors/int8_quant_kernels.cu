#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

#include <cmath>

#include "../../dispatch_utils.h"
#include "../vectorization_utils.cuh"

#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>

static inline __device__ int8_t float_to_int8_rn(float x) {
  // CUDA path
  // uint32_t dst;
  // asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  // return reinterpret_cast<const int8_t&>(dst);
  int32_t dst;
  dst = __float2int_rn(x);
  dst = min(dst, 127);
  dst = max(dst, -127);
  return reinterpret_cast<const int8_t&>(dst);
}

static inline __device__ int32_t float_to_int32_rn(float x) {
  // CUDA path
  static constexpr auto i32_min = std::numeric_limits<int32_t>::min();
  static constexpr auto i32_min_f = static_cast<float>(i32_min);
  static constexpr auto i32_max = std::numeric_limits<int32_t>::max();
  static constexpr auto i32_max_f = static_cast<float>(i32_max);
  x = min(x, i32_max_f);
  x = max(x, i32_min_f);
  return __float2int_rn(x);
}

static inline __device__ int8_t int32_to_int8(int32_t x) {
  // CUDA path
  static constexpr auto i8_min =
      static_cast<int32_t>(std::numeric_limits<int8_t>::min());
  static constexpr auto i8_max =
      static_cast<int32_t>(std::numeric_limits<int8_t>::max());

  // saturate
  int32_t dst = std::clamp(x, i8_min, i8_max);
  return static_cast<int8_t>(dst);
}

namespace vllm {

template <typename scalar_t, typename scale_t>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;
  const float scale = *scale_ptr;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(static_cast<float>(src) / scale);
      });
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void static_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    const scale_t* scale_ptr, const azp_t* azp_ptr, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;
  const float scale = *scale_ptr;
  const azp_t azp = *azp_ptr;
  const float inv_s = 1.0f / scale;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        const auto v = static_cast<float>(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

template <typename scalar_t, typename scale_t>
__global__ void dynamic_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  // calculate for absmax
  float thread_max = 0.f;
  vectorize_read_with_alignment<16>(
      row_in, hidden_size, tid, stride, [&] __device__(const scalar_t& src) {
        const float v = fabsf(static_cast<float>(src));
        thread_max = fmaxf(thread_max, v);
      });

  using BlockReduce = hipcub::BlockReduce<float, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;
  float block_max = BlockReduce(tmp).Reduce(thread_max, hipcub::Max{}, blockDim.x);
  __shared__ float absmax;
  if (tid == 0) {
    absmax = block_max;
    scale_out[blockIdx.x] = absmax / 127.f;
  }
  __syncthreads();

  float inv_s = (absmax == 0.f) ? 0.f : 127.f / absmax;

  // 2. quantize
  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        dst = float_to_int8_rn(static_cast<float>(src) * inv_s);
      });
}

// MinMax structure to hold min and max values in one go
struct MinMax {
  float min, max;

  __host__ __device__ MinMax()
      : min(std::numeric_limits<float>::max()),
        max(std::numeric_limits<float>::lowest()) {}

  __host__ __device__ explicit MinMax(float v) : min(v), max(v) {}

  // add a value to the MinMax
  __host__ __device__ MinMax& operator+=(float v) {
    min = fminf(min, v);
    max = fmaxf(max, v);
    return *this;
  }

  // merge two MinMax objects
  __host__ __device__ MinMax& operator&=(const MinMax& other) {
    min = fminf(min, other.min);
    max = fmaxf(max, other.max);
    return *this;
  }
};

__host__ __device__ inline MinMax operator+(MinMax a, float v) {
  return a += v;
}
__host__ __device__ inline MinMax operator&(MinMax a, const MinMax& b) {
  return a &= b;
}

template <typename scalar_t, typename scale_type, typename VT, typename VT1>
__global__ void dynamic_scaled_int8_quant_kernel_sreg_opt(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size, int blockDim_x) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;
  scalar_t absmax_val = static_cast<scalar_t>(0.0f);
  float const zero = 0.0f;
  constexpr int N = sizeof(VT) / sizeof(scalar_t);
  scalar_t reg_src0[N];
  scalar_t const* ptr_input = input + token_idx * hidden_size;
  int reg_length = blockDim_x * N;
  int length = min(hidden_size, reg_length);
  int index = tid * N;
  if (index < length) {
    *(VT*)reg_src0 = *(VT*)(ptr_input + index);
#pragma unroll N
    for (int i = 0; i < N; i++) {
      scalar_t val = abs(reg_src0[i]);
      absmax_val = max(absmax_val, val);
    }
  }

  using BlockReduce = hipcub::BlockReduce<scalar_t, 512>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim_x);
  __shared__ scale_type block_absmax_val;
  if (tid == 0) {
    block_absmax_val = static_cast<scale_type>(block_absmax_val_maybe);
    scale[token_idx] = static_cast<scale_type>(block_absmax_val / 127.0f);
  }
  __syncthreads();
  float const tmp_scale = 127.0f / block_absmax_val;
  int8_t* ptr_output = out + token_idx * hidden_size;
  if (index < length) {
    VT1 vdst;
    int8_t* ptr_reg = (int8_t*)&vdst;
#pragma unroll N
    for (int i = 0; i < N; i++) {
      ptr_reg[i] =
          float_to_int8_rn(static_cast<float>(reg_src0[i]) * tmp_scale);
    }
    *(VT1*)(ptr_output + index) = vdst;
  }
}

template <typename scalar_t, typename scale_type, typename VT, typename VT1>
__global__ void dynamic_scaled_int8_quant_kernel_reg_opt(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size, int blockDim_x) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;
  scalar_t absmax_val = static_cast<scalar_t>(0.0f);
  float const zero = 0.0f;
  constexpr int N = sizeof(VT) / sizeof(scalar_t);
  scalar_t reg_src0[N];
  scalar_t reg_src1[N];
  scalar_t const* ptr_input = input + token_idx * hidden_size;
  int reg_length = 2 * blockDim_x * N;
  int length = min(hidden_size, reg_length);
  int index = 2 * tid * N;
  if (index < length) {
    *(VT*)reg_src0 = *(VT*)(ptr_input + index);
    *(VT*)reg_src1 = *(VT*)(ptr_input + index + N);
#pragma unroll N
    for (int i = 0; i < N; i++) {
      scalar_t val = abs(reg_src0[i]);
      absmax_val = max(val, absmax_val);
      val = abs(reg_src1[i]);
      absmax_val = max(val, absmax_val);
    }
  }

  using BlockReduce = hipcub::BlockReduce<scalar_t, 512>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  scalar_t const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim_x);
  __shared__ scale_type block_absmax_val;
  if (tid == 0) {
    block_absmax_val = static_cast<scale_type>(block_absmax_val_maybe);
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();
  float const tmp_scale = 127.0f / block_absmax_val;
  int8_t* ptr_output = out + token_idx * hidden_size;
  if (index < length) {
    VT1 vdst;
    int8_t* ptr_reg = (int8_t*)&vdst;
    constexpr int ON = 2 * N;
#pragma unroll N
    for (int i = 0; i < N; i++) {
      ptr_reg[i] =
          float_to_int8_rn(static_cast<float>(reg_src0[i]) * tmp_scale);
    }
    ptr_reg = ptr_reg + N;
#pragma unroll N
    for (int i = 0; i < N; i++) {
      ptr_reg[i] =
          float_to_int8_rn(static_cast<float>(reg_src1[i]) * tmp_scale);
    }
    *(VT1*)(ptr_output + index) = vdst;
  }
}

template <typename scalar_t, typename scale_type, typename VT, typename VT1>
__global__ void dynamic_scaled_int8_quant_kernel_sm_opt(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size, int blockDim_x) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  float const zero = 0.0f;
  constexpr int N = sizeof(VT) / sizeof(scalar_t);
  int stride = blockDim_x * N;
  __shared__ float sm_buffer[8064];
  scalar_t const* ptr_input = input + token_idx * hidden_size;
  for (int i = tid * N; i < hidden_size; i += stride) {
    VT vsrc = *(VT*)(ptr_input + i);
    scalar_t* ptr_src = (scalar_t*)&vsrc;
    float* ptr_sm_buffer = sm_buffer + i;
#pragma unroll N
    for (int j = 0; j < N; j++) {
      float val = static_cast<float>(ptr_src[j]);
      ptr_sm_buffer[j] = val;
      val = val > zero ? val : -val;
      absmax_val = val > absmax_val ? val : absmax_val;
    }
  }
  using BlockReduce = hipcub::BlockReduce<float, 512>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }

  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  int8_t* ptr_output = out + token_idx * hidden_size;
  for (int i = tid * N; i < hidden_size; i += stride) {
    VT1 vdst;
    int8_t* ptr_reg = (int8_t*)&vdst;
    float* ptr_sm_buffer = sm_buffer + i;
#pragma unroll N
    for (int j = 0; j < N; j++) {
      ptr_reg[j] = float_to_int8_rn(ptr_sm_buffer[j] * tmp_scale);
    }
    *(VT1*)(ptr_output + i) = vdst;
  }
}

template <typename scalar_t, typename scale_type, typename VT, typename VT1>
__launch_bounds__(1024) __global__ void dynamic_scaled_int8_quant_kernel_opt(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size, const int blockDim_x) {
  constexpr int N = sizeof(VT) / sizeof(scalar_t);
  int const tid = threadIdx.x * N;
  int64_t const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  int stride = blockDim_x * N;
  const scalar_t* ptr_input = input + token_idx * hidden_size;

  for (int i = tid; i < hidden_size; i += stride) {
    VT vsrc = *(VT*)(ptr_input + i);
    scalar_t* ptr_src = (scalar_t*)&vsrc;
#pragma unroll N
    for (int j = 0; j < N; j++) {
      float val = static_cast<float>(ptr_src[j]);
      val = val > 0 ? val : -val;
      absmax_val = val > absmax_val ? val : absmax_val;
    }
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  int8_t* ptr_output = out + token_idx * hidden_size;
  for (int i = tid; i < hidden_size; i += stride) {
    VT vsrc = *(VT*)(ptr_input + i);
    VT1 vdst;
    scalar_t* ptr_src = (scalar_t*)&vsrc;
    int8_t* ptr_dst = (int8_t*)&vdst;
#pragma unroll N
    for (int j = 0; j < N; ++j) {
      ptr_dst[j] = float_to_int8_rn(static_cast<float>(ptr_src[j]) * tmp_scale);
    }
    *(VT1*)(ptr_output + i) = vdst;
  }
}

template <typename scalar_t, typename scale_t, typename azp_t>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ output,
    scale_t* scale_out, azp_t* azp_out, const int hidden_size) {
  const int tid = threadIdx.x;
  const int stride = blockDim.x;
  const int64_t token_idx = blockIdx.x;

  // Must be performed using 64-bit math to avoid integer overflow.
  const scalar_t* row_in = input + token_idx * hidden_size;
  int8_t* row_out = output + token_idx * hidden_size;

  // 1. calculate min & max
  MinMax thread_mm;
  vectorize_read_with_alignment<16>(row_in, hidden_size, tid, stride,
                                    [&] __device__(const scalar_t& src) {
                                      thread_mm += static_cast<float>(src);
                                    });

  using BlockReduce = hipcub::BlockReduce<MinMax, 256>;
  __shared__ typename BlockReduce::TempStorage tmp;

  MinMax mm = BlockReduce(tmp).Reduce(
      thread_mm,
      [] __device__(MinMax a, const MinMax& b) {
        a &= b;
        return a;
      },
      blockDim.x);

  __shared__ float scale_sh;
  __shared__ azp_t azp_sh;
  if (tid == 0) {
    float s = (mm.max - mm.min) / 255.f;
    float zp = nearbyintf(-128.f - mm.min / s);  // round-to-even
    scale_sh = s;
    azp_sh = azp_t(zp);
    scale_out[blockIdx.x] = s;
    azp_out[blockIdx.x] = azp_sh;
  }

  __syncthreads();

  const float inv_s = 1.f / scale_sh;
  const azp_t azp = azp_sh;

  // 2. quantize
  vectorize_with_alignment<16>(
      row_in, row_out, hidden_size, tid, stride,
      [=] __device__(int8_t& dst, const scalar_t& src) {
        const auto v = static_cast<float>(src) * inv_s;
        dst = int32_to_int8(float_to_int32_rn(v) + azp);
      });
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale,
                              std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);
  TORCH_CHECK(!azp || azp->numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::static_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), hidden_size);
        } else {
          vllm::static_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scale.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales, std::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scales.is_contiguous());
  TORCH_CHECK(!azp || azp->is_contiguous());

  int const hidden_size = input.size(-1);
  int64_t const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 256));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          int n = 16 / sizeof(scalar_t);
          if (hidden_size <= 4096 && ((hidden_size & (n - 1)) == 0) && n == 8) {
            int64_t gridsize = num_tokens;
            int blocksize = 512;
            vllm::dynamic_scaled_int8_quant_kernel_sreg_opt<scalar_t, float,
                                                            float4, float2>
                <<<gridsize, blocksize, 0, stream>>>(
                    input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                    scales.data_ptr<float>(), hidden_size, blocksize);
          } else if (hidden_size > 4096 && hidden_size <= 8192 &&
                     ((hidden_size & (2 * n - 1)) == 0) && n == 8) {
            int64_t gridsize = num_tokens;
            int blocksize = 512;
            vllm::dynamic_scaled_int8_quant_kernel_reg_opt<scalar_t, float,
                                                           float4, float4>
                <<<gridsize, blocksize, 0, stream>>>(
                    input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                    scales.data_ptr<float>(), hidden_size, blocksize);
          } else if (hidden_size <= 8064 && (hidden_size & (n - 1)) == 0 &&
                     n == 8) {
            int64_t gridsize = num_tokens;
            int blocksize = 512;
            vllm::dynamic_scaled_int8_quant_kernel_sm_opt<scalar_t, float,
                                                          float4, float2>
                <<<gridsize, blocksize, 0, stream>>>(
                    input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                    scales.data_ptr<float>(), hidden_size, blocksize);
          } else if (hidden_size > 8064 &&
                     ((hidden_size & (n - 1)) == 0 && n == 8)) {
            int blocksize = 1024;
            vllm::dynamic_scaled_int8_quant_kernel_opt<scalar_t, float, float4,
                                                       float2>
                <<<grid, blocksize, 0, stream>>>(
                    input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                    scales.data_ptr<float>(), hidden_size, blocksize);
          } else {
            vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
                <<<grid, block, 0, stream>>>(
                    input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                    scales.data_ptr<float>(), hidden_size);
          }
        } else {
          vllm::dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}